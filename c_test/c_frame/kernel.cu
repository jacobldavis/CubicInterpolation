#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2025 Jacob Davis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 * 
 * The purpose of this program is to control profiling
 * different frameworks for a cubic interpolation.
 */
#include <hip/hip_runtime.h>
#include "kernel.h"

__global__ void cubic_interp_eval(int c, cubic_interp* dev_interp, double* dev_t) {
    // Sets initial index and other values to compute
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int len = dev_interp->length;
    double f = dev_interp->f;
    double t0 = dev_interp->t0;
    double xmin = 0.0, xmax = dev_interp->length - 1.0;

    // Performs cubic interpolation
    while (idx < c) {
        double x = dev_t[idx];
        x = f * x + t0;
        x = fmin(fmax(x, xmin), xmax);

        int ix = (int)x;
        x -= ix;

        const double *a = dev_interp->a[ix];
        dev_t[idx] = (x * (x * (x * a[0] + a[1]) + a[2]) + a[3]);

        idx += stride;
    }
}

extern void test_all_cubic_cuda(double **values, FILE *fp) 
{
    printf("Testing CUDA cubic\n");
    fprintf(fp, "Data,Iterations,Time\n");

    // Iterates through the test for each size in n_values
    for (int i = 0; i < n_values_size; i++) {
        // Initializes cubic_interp and copies to the GPU
        cubic_interp *interp = cubic_interp_init(values[i], n_values[i], -1, 1);
        double (*host_a)[4] = interp->a;
        int len = interp->length;

        double (*dev_a)[4];
        hipMalloc(&dev_a, len * sizeof(double[4]));
        hipMemcpy(dev_a, host_a, len * sizeof(double[4]), hipMemcpyHostToDevice);

        cubic_interp h_interp_dev;
        h_interp_dev.length = len;
        h_interp_dev.f = interp->f;
        h_interp_dev.t0 = interp->t0;
        h_interp_dev.a = dev_a;

        cubic_interp *dev_interp;
        hipMalloc(&dev_interp, sizeof(cubic_interp));
        hipMemcpy(dev_interp, &h_interp_dev, sizeof(cubic_interp), hipMemcpyHostToDevice);

        // Iterates through the interpolation with varying evaluation counts
        int c = 10000;
        for (int m = 1; m < 5; m++) {
            // Precomputes random values and other relevant vars for CUDA
            double* t = (double*)malloc(c * sizeof(double));
            for (int k = 0; k < c; k++) {
                t[k] = rand() * 100;
            }
            int threadsPerBlock = 1028;
            int blocksPerGrid = int((c+threadsPerBlock-1)/threadsPerBlock);
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            float elapsedTime;

            // Copies t to the GPU
            double* dev_t;
            hipMalloc( (void**)&dev_t, c * sizeof(double));
            hipMemcpy(dev_t, t, c * sizeof(double), hipMemcpyHostToDevice);

            // Performs benchmark and records time
            hipEventRecord(start, 0);
            cubic_interp_eval<<<blocksPerGrid,threadsPerBlock>>>(c, dev_interp, dev_t);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start, stop);
            hipMemcpy(t, dev_t, c*sizeof(double), hipMemcpyDeviceToHost); // include or exclude from timing?
            printf("Time for size %d and iterations %d is %lf\n", n_values[i], c, elapsedTime);
            fprintf(fp, "%d,%d,%lf\n", n_values[i], c, elapsedTime);

            // Frees t, dev_t, and events
            hipFree(dev_t);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            free(t);

            c *= 10;
        }
        // Frees interp related variables
        hipFree(dev_interp);
        hipFree(dev_a);
        cubic_interp_free(interp);
        printf("\n");
    }
}